
#include <hip/hip_runtime.h>
#include <string.h>
#include <iostream>

#include <time.h>
#include <stdlib.h>

#define WIDTH 2048
#define HEIGHT 2048
#define NUM_TESTS 1000

#define CPU 1
#define GPU 0

#define CHECKBOARD_BLOCK_MAX_WIDTH  1
#define CHECKBOARD_BLOCK_MAX_HEIGHT 256

class Matrix {
private:
	const unsigned int width_;
	const unsigned int height_;
	const bool isCPU_;
	
	int *smart_ptr_counter_; // num of instances of Matrix which share data_
	float *data_;

public:
	
	Matrix(const unsigned int &width, const unsigned int &height, const bool &isCPU) : width_(width), height_(height), isCPU_(isCPU) {
		smart_ptr_counter_ = new int(1);
		if (isCPU_)
			data_ = new float[width_ * height_];
		else
			hipMalloc(&data_, width_ * height_ * sizeof(float));
	}
	
	Matrix(const Matrix &m) : width_(m.width_), height_(m.height_), data_(m.data_), isCPU_(m.isCPU_), smart_ptr_counter_(m.smart_ptr_counter_) {
		(*smart_ptr_counter_) += 1;
	}
	
	Matrix(const Matrix &m, const bool &isCPU) : width_(m.width_), height_(m.height_), isCPU_(isCPU) {
		smart_ptr_counter_ = new int(1);
		if (isCPU_) {
			data_ = new float[width_ * height_];
			if (m.isCPU_)
				memcpy(data_, m.data_, width_ * height_ * sizeof(float));
			else
				hipMemcpy(data_, m.data_, width_ * height_ * sizeof(float), hipMemcpyDeviceToHost);
		} else {
			hipMalloc(&data_, width_ * height_ * sizeof(float));
			if (m.isCPU_)
				hipMemcpy(data_, m.data_, width_ * height_ * sizeof(float), hipMemcpyHostToDevice);
			else
				hipMemcpy(data_, m.data_, width_ * height_ * sizeof(float), hipMemcpyDeviceToDevice);
		}
	}
	
	~Matrix() {
		if (*smart_ptr_counter_ > 1) {
			(*smart_ptr_counter_) -= 1;
			return;
		}
		
		delete smart_ptr_counter_;
		
		if (isCPU_)
			delete [] data_;
		else
			hipFree(data_);
	}
	
	void memsetZero() {
		if (isCPU_)
			memset(data_, 0, width_ * height_ * sizeof(float));
		else
			hipMemset(data_, 0, width_ * height_ * sizeof(float));
	}
	
	class MatrixRow { // cf. http://stackoverflow.com/questions/3755111/how-do-i-define-a-double-brackets-double-iterator-operator-similar-to-vector-of
		const Matrix &m_;
		const unsigned int x_;
		
		public:
			__device__ __host__ MatrixRow(const Matrix &m, const unsigned int &x) : m_(m), x_(x) {}
			
			/*
				WARNING
				Prefer using .get(x, y)
				for performances
			*/
			__device__ __host__ float& operator[](const unsigned int &y) const {
				return m_.data_[x_ * m_.width_ + y];
			}
	};
	
	/*
		Return the ith element of the matrix
		Equivalent to:
			(*this)[row][col]
		With:
			row = i / width_
			col = i % width_
		
		More efficient to retrieve an element
		It avoids having to create an instance of MatrixRow
		
		(*this).get(x * width_ + y) is more efficient than (*this)[x][y]
	*/
	__device__ __host__ float& get(const unsigned int &i) const {
		return data_[i];
	}
	
	/*
		More efficient than [][] to retrieve an element
		(*this).get(x, y) is more efficient than (*this)[x][y]
	*/
	__device__ __host__ float& get(const unsigned int &i, const unsigned int &j) const {
		return data_[i * width_ + j];
	}
		
	__device__ __host__ MatrixRow operator[](const unsigned int &x) const {
		return MatrixRow(*this, x);
	}
	
	__device__ __host__ unsigned int getWidth() const { return width_; }
	__device__ __host__ unsigned int getHeight() const { return height_; }
	__device__ __host__ bool isCPU() const { return isCPU_; }
	
	void print() {
		std::cout << "[" << std::endl;
		for (unsigned int i(0) ; i != height_ ; i++) {
			for (unsigned int j(0) ; j != width_ ; j++) {
				std::cout << " " << this->get(i, j);
			}
			std::cout << std::endl;
		}
		std::cout << "]" << std::endl;
	}
};

class Vector {
private:
	const unsigned int size_;
	const bool isCPU_;
	
	int *smart_ptr_counter_; // num of instances of Vector which share data_
	float *data_;

public:
	Vector(const unsigned int &size, const bool &isCPU) : size_(size), isCPU_(isCPU) {
		smart_ptr_counter_ = new int(1);
		if (isCPU_)
			data_ = new float[size_];
		else
			hipMalloc(&data_, size_ * sizeof(float));
	}
	
	Vector(const Vector &v) : size_(v.size_), data_(v.data_), isCPU_(v.isCPU_), smart_ptr_counter_(v.smart_ptr_counter_) {
		(*smart_ptr_counter_) += 1;
	}
	
	Vector(const Vector &v, const bool &isCPU) : size_(v.size_), isCPU_(isCPU) {
		smart_ptr_counter_ = new int(1);
		if (isCPU_) {
			data_ = new float[size_];
			if (v.isCPU_)
				memcpy(data_, v.data_, size_ * sizeof(float));
			else
				hipMemcpy(data_, v.data_, size_ * sizeof(float), hipMemcpyDeviceToHost);
		} else {
			hipMalloc(&data_, size_ * sizeof(float));
			if (v.isCPU_)
				hipMemcpy(data_, v.data_, size_ * sizeof(float), hipMemcpyHostToDevice);
			else
				hipMemcpy(data_, v.data_, size_ * sizeof(float), hipMemcpyDeviceToDevice);
		}
	}
	
	~Vector() {
		if(*smart_ptr_counter_ > 1) {// cuda-kernel constructs a copy of the object and then call its destructor
			(*smart_ptr_counter_) -= 1;
			return;
		}
		
		delete smart_ptr_counter_;
		
		if (isCPU_)
			delete [] data_;
		else
			hipFree(data_);
	}
	
	void memsetZero() {
		if (isCPU_)
			memset(data_, 0, size_ * sizeof(float));
		else
			hipMemset(data_, 0, size_ * sizeof(float));
	}
	
	__device__ __host__ float& get(const unsigned int &x) const {
		return data_[x];
	}
	
	__device__ __host__ float& operator[](const unsigned int &x) const {
		return data_[x];
	}
	
	__device__ __host__ unsigned int getSize() const { return size_; }
	__device__ __host__ bool isCPU() const { return isCPU_; }
	
	void print() {
		std::cout << "[";
		for (unsigned int j(0) ; j != size_ ; j++) {
			std::cout << " " << (*this)[j];
		}
		std::cout << " ]" << std::endl;
	}
};

/*
	Serial implementation of Matrix-Vector product
	for CPU
	
	TODO check: m.getWidth() = v.getSize()
*/
Vector productMatrixVectorCPU(const Matrix &m, const Vector &v) {
	Vector r(m.getHeight(), CPU);
	
	unsigned int id(0);
	for (unsigned int i(0) ; i != m.getHeight() ; i++) {
		r[i] = 0;
		for (unsigned int j(0) ; j != m.getWidth() ; j++) {
			r[i] += m.get(id) * v[j]; //r[i] += m[i][j] * v[j];
			id++;
		}
	}
	return r;
}

__global__ void productMatrixVectorGPU_naive_kernel(const Matrix d_m, const Vector d_v, Vector d_r) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (i >= d_m.getHeight() || j >= d_m.getWidth())
		return;
	
	atomicAdd(&d_r[i], d_m.get(i, j) * d_v[j]);
}

/*
	Parallel implementation of Matrix-Vector product
	for GPU
	
	Naïve implementation:
		Checkboard partitioning
	
	TODO check: m.getWidth() = v.getSize()
*/
Vector productMatrixVectorGPU_naive(const Matrix &h_m, const Vector &h_v) {
	Matrix d_m(h_m, GPU);
	Vector d_v(h_v, GPU);
	
	Vector d_r(h_m.getHeight(), GPU);
	d_r.memsetZero();
	
	const dim3 num_threads(CHECKBOARD_BLOCK_MAX_HEIGHT, CHECKBOARD_BLOCK_MAX_WIDTH, 1);
	const dim3 num_blocks((d_m.getHeight() + CHECKBOARD_BLOCK_MAX_HEIGHT -1)/CHECKBOARD_BLOCK_MAX_HEIGHT, (d_m.getWidth() + CHECKBOARD_BLOCK_MAX_WIDTH -1)/CHECKBOARD_BLOCK_MAX_WIDTH, 1);
	productMatrixVectorGPU_naive_kernel<<<num_blocks, num_threads>>>(d_m, d_v, d_r);
	
	Vector h_r(d_r, CPU);
	return h_r;
}

__global__ void productMatrixVectorGPU_shared_kernel(const Matrix d_m, const Vector d_v, Vector d_r) {
	extern __shared__ float block_result[]; // best value: blockDim.x + blockDim.y
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	block_result[threadIdx.x] = 0;
	block_result[blockDim.x + threadIdx.y] = d_v[j];
	__syncthreads();
	
	if (i >= d_m.getHeight() || j >= d_m.getWidth())
		return;
	
	atomicAdd(&block_result[threadIdx.x], d_m.get(i, j) * block_result[blockDim.x + threadIdx.y]);
	__syncthreads();
	
	if (threadIdx.y == 0)
		atomicAdd(&d_r[i], block_result[threadIdx.x]);
}

/*
	Parallel implementation of Matrix-Vector product
	for GPU
	
	Shared implementation:
		Checkboard partitioning
		Shared memory
	
	TODO check: m.getWidth() = v.getSize()
*/
Vector productMatrixVectorGPU_shared(const Matrix &h_m, const Vector &h_v) {
	Matrix d_m(h_m, GPU);
	Vector d_v(h_v, GPU);
	
	Vector d_r(h_m.getHeight(), GPU);
	d_r.memsetZero();
	
	const dim3 num_threads(CHECKBOARD_BLOCK_MAX_HEIGHT, CHECKBOARD_BLOCK_MAX_WIDTH, 1);
	const dim3 num_blocks((d_m.getHeight() + CHECKBOARD_BLOCK_MAX_HEIGHT -1)/CHECKBOARD_BLOCK_MAX_HEIGHT, (d_m.getWidth() + CHECKBOARD_BLOCK_MAX_WIDTH -1)/CHECKBOARD_BLOCK_MAX_WIDTH, 1);
	productMatrixVectorGPU_shared_kernel<<<num_blocks, num_threads, CHECKBOARD_BLOCK_MAX_HEIGHT + CHECKBOARD_BLOCK_MAX_WIDTH>>>(d_m, d_v, d_r);
	
	Vector h_r(d_r, CPU);
	return h_r;
}

int main(int argc, char **argv) {
	srand(time(NULL));
	
	Matrix h_m(WIDTH, HEIGHT, CPU);
	for (unsigned int i(0) ; i != HEIGHT ; i++) {
		for (unsigned int j(0) ; j != WIDTH ; j++) {
			h_m[i][j] = rand() % 10;
		}
	}
	//h_m.print();
	
	Vector h_v(WIDTH, CPU);
	for (unsigned int j(0) ; j != WIDTH ; j++) {
		h_v[j] = rand() % 10;
	}
	//h_v.print();
	
	clock_t t_chrono;
	t_chrono = clock();
	for (int i(0) ; i != NUM_TESTS ; i++) {
		productMatrixVectorCPU(h_m, h_v);
	}
	t_chrono = clock() - t_chrono;
	std::cout << "Measured time for <CPU>: " << ((float) t_chrono)/CLOCKS_PER_SEC << "s" << std::endl;
	
	t_chrono = clock();
	for (int i(0) ; i != NUM_TESTS ; i++) {
		productMatrixVectorGPU_naive(h_m, h_v);
	}
	t_chrono = clock() - t_chrono;
	std::cout << "Measured time for <GPU naive>: " << ((float) t_chrono)/CLOCKS_PER_SEC << "s" << std::endl;
	
	t_chrono = clock();
	for (int i(0) ; i != NUM_TESTS ; i++) {
		productMatrixVectorGPU_shared(h_m, h_v);
	}
	t_chrono = clock() - t_chrono;
	std::cout << "Measured time for <GPU shared>: " << ((float) t_chrono)/CLOCKS_PER_SEC << "s" << std::endl;
	
	return 0;
}

